#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>



__global__ void print_warp_details() {
	int gid = blockIdx.y * gridDim.x * blockIdx.x + blockDim.x * blockIdx.x + threadIdx.x;
	
	int warp_id = threadIdx.x / 32;

	// block index
	int gbid = blockIdx.y * gridDim.x + blockIdx.x;

	printf("tid: %d, bid.x: %d, bid.y: %d, gid: %d, warp_id: %d, gbid: %d\n", threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
}

int main() {

	dim3 block(42);
	dim3 grid(2, 2);

	print_warp_details << <grid, block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return EXIT_SUCCESS;
}