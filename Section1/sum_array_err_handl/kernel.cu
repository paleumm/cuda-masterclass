#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cstring>

#include "cuda_common.cuh"

void compare_arrays(int* a, int* b, int size);

__global__ void sum_array(int* a, int* b, int* c, int size) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size) {
		c[gid] = a[gid] + b[gid];
	}
}

void sum_array_cpu(int* a, int* b, int* c, int size) {
	for (int i = 0; i < size; i++) {
		c[i] = a[i] + b[i];
	}
}

int main() {
	int size = 1 << 25;
	int byte_size = size * sizeof(int);

	int block_size = 1024;

	hipError_t err;

	int* h_a, * h_b, * h_c, * results;

	h_a = (int*)malloc(byte_size);
	h_b = (int*)malloc(byte_size);
	h_c = (int*)malloc(byte_size);
	results = (int*)malloc(byte_size);

	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++) {
		h_a[i] = (int)(rand() & 0xff);
	}

	for (int i = 0; i < size; i++) {
		h_b[i] = (int)(rand() & 0xff);
	}

	// sum using cpu
	clock_t cpu_start, cpu_end;
	cpu_start = clock();
	sum_array_cpu(h_a, h_b, h_c, size);
	cpu_end = clock();

	memset(results, 0, byte_size);

	int* d_a, * d_b, * d_c;

	gpuAssert(hipMalloc((int**)&d_a, byte_size));
	gpuAssert(hipMalloc((int**)&d_b, byte_size));
	gpuAssert(hipMalloc((int**)&d_c, byte_size));
	
	dim3 block(block_size);
	dim3 grid((size / block.x) + 1);

	clock_t htod_start, htod_end;
	htod_start = clock();
	hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice);
	htod_end = clock();

	clock_t gpu_start, gpu_end;
	gpu_start = clock();
	sum_array << <grid, block >> > (d_a, d_b, d_c, size);
	hipDeviceSynchronize();
	gpu_end = clock();

	clock_t dtoh_start, dtoh_end;
	dtoh_start = clock();
	hipMemcpy(results, d_c, byte_size, hipMemcpyDeviceToHost);
	dtoh_end = clock();

	// array comparison
	compare_arrays(h_c, results, size);

	printf("Sum array on CPU execution time : %4.6f\n", 
		(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));

	printf("Sum array on GPU execution time : %4.6f\n",
		(double)((double)(gpu_end - gpu_start) / CLOCKS_PER_SEC));

	printf("htod mem transfer time : %4.6f\n",
		(double)((double)(htod_end - htod_start) / CLOCKS_PER_SEC));

	printf("dtoh mem transfer time : %4.6f\n",
		(double)((double)(dtoh_end - dtoh_start) / CLOCKS_PER_SEC));

	printf("Total GPU execution time : %4.6f\n",
		(double)((double)(dtoh_end - htod_start) / CLOCKS_PER_SEC));

	/*for (int i = 0; i < size; i++) {
		if (i % 128 == 0) printf("\n\n");

		printf("%d ", results[i]);
	}*/

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(h_a);
	free(h_b);
	free(results);

	hipDeviceReset();
	return 0;
}

void compare_arrays(int* a, int* b, int size) {
	for (int i = 0; i < size; i++) {
		if (a[i] != b[i]) {
			printf("Arrays are different\n");
			return;
		}
	}
	printf("Arrays are same\n");
}