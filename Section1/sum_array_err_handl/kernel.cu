#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdlib.h>
#include <time.h>

#include <cstring>

#include "cuda_common.cuh"

void compare_arrays(int* a, int* b, int size);

__global__ void sum_array(int* a, int* b, int* c, int size) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size) {
		c[gid] = a[gid] + b[gid];
	}
}

void sum_array_cpu(int* a, int* b, int* c, int size) {
	for (int i = 0; i < size; i++) {
		c[i] = a[i] + b[i];
	}
}

int main() {
	int size = 1 << 25;
	int byte_size = size * sizeof(int);

	int block_size = 1024;

	hipError_t err;

	int* h_a, * h_b, * h_c, * results;

	h_a = (int*)malloc(byte_size);
	h_b = (int*)malloc(byte_size);
	h_c = (int*)malloc(byte_size);
	results = (int*)malloc(byte_size);

	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++) {
		h_a[i] = (int)(rand() & 0xff);
	}

	for (int i = 0; i < size; i++) {
		h_b[i] = (int)(rand() & 0xff);
	}

	sum_array_cpu(h_a, h_b, h_c, size);

	memset(results, 0, byte_size);

	int* d_a, * d_b, * d_c;

	gpuAssert(hipMalloc((int**)&d_a, byte_size));
	gpuAssert(hipMalloc((int**)&d_b, byte_size));
	gpuAssert(hipMalloc((int**)&d_c, byte_size));
	
	hipMemcpy(d_a, h_a, byte_size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, byte_size, hipMemcpyHostToDevice);

	dim3 block(block_size);
	dim3 grid((size / block.x) + 1);

	sum_array << <grid, block >> > (d_a, d_b, d_c, size);
	hipDeviceSynchronize();

	hipMemcpy(results, d_c, byte_size, hipMemcpyDeviceToHost);

	// array comparison
	compare_arrays(h_c, results, size);

	/*for (int i = 0; i < size; i++) {
		if (i % 128 == 0) printf("\n\n");

		printf("%d ", results[i]);
	}*/

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(h_a);
	free(h_b);
	free(results);

	hipDeviceReset();
	return 0;
}

void compare_arrays(int* a, int* b, int size) {
	for (int i = 0; i < size; i++) {
		if (a[i] != b[i]) {
			printf("Arrays are different\n");
			return;
		}
	}
	printf("Arrays are same\n");
}